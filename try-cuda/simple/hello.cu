
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void say_hello_gpu(void) {
	printf("hello from gpu %d, %d\n", threadIdx.x, blockIdx.x);
}

int main(void) {

	printf("hello from cpu\n");
	say_hello_gpu<<<2,2>>>();
	hipDeviceSynchronize();
	return 0;
}
